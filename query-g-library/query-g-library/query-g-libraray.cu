#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "query-g-libraray.h"
#include "book.h"

namespace qgl {
	QGLibraray::QGLibraray() {
	}

	QGLibraray::~QGLibraray() {

	}

	/* kernel to increase book's id by given ammount */
	__global__ void increese_book_id(Book *book, int ammount)
	{
		// Get our global thread ID
		int id = blockIdx.x*blockDim.x + threadIdx.x;

		book->author_id += ammount;
		//book->
	}

	void QGLibraray::increse_book_id(Book *book, int amount) {

		// host
		Book *h_input;

		// device
		Book *d_input;

		// determine size in bytes for later allocation
		size_t size = sizeof(Book);

		// alocate memory on host
		h_input = book;

		// alocate memory on device
		hipMalloc(&d_input, size);

		// copy from host to device
		hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

		// execute kernel
		increese_book_id << <1, 1 >> >(d_input, amount);

		// copy from device to host
		hipMemcpy(h_input, d_input, size, hipMemcpyDeviceToHost);

		// release memory
		hipFree(d_input);
	}
}
